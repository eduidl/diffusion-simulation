#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "diffusion.cuh"
#include "bmp.h"

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <iomanip>
#include <sstream>

#define blockDim_x 256
#define blockDim_y 8 

#define CUDA_SAFE_CALL(call) \
{ \
  const hipError_t error = call; \
  if (error != hipSuccess) { \
    fprintf(stderr, "[Error]: %s:%d, ", __FILE__, __LINE__); \
    fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error)); \
    exit(1); \
  } \
}

namespace diffusion {

template <typename T>
__global__ void diffusion2d_0(T *f, T *f_tmp,
                            const size_t nx, const size_t ny,
                            const T c0, const T c1, const T c2) {
  const auto jx = blockDim.x * blockIdx.x + threadIdx.x;
  const auto jy = blockDim.y * blockIdx.y + threadIdx.y;

  const auto j = nx * jy + jx;
  const auto fcc = f[j];

  const auto fcw = (jx == 0) ? fcc : f[j - 1];
  const auto fce = (jx == nx - 1) ? fcc : f[j + 1];

  const auto fcn = (jy == 0) ? fcc : f[j - nx];
  const auto fcs = (jy == ny - 1) ? fcc : f[j + nx];

  f_tmp[j] = c0 * (fce + fcw) + c1 * (fcn + fcs) + c2 * fcc;
}

template <typename T>
__global__ void diffusion2d(T *f, T *f_tmp,
                            const size_t nx, const size_t ny,
                            const T c0, const T c1, const T c2) {
  __shared__ T fs[blockDim_x + 2];

  const auto jy = blockDim_y * blockIdx.y;
  auto j = nx * jy + blockDim.x * blockIdx.x + threadIdx.x;

  auto f1 = f[j];
  auto f0 = (blockIdx.y == 0) ? f1 : f[j - nx];
  j += nx;
  T f2;

#pragma unroll
  for (auto jy = 0; jy < blockDim_y; jy++) {
    f2 = (blockIdx.y == gridDim.y - 1) ? f1 : f[j];
    fs[threadIdx.x + 1] = f1;
     
    if (threadIdx.x == 0) {
      fs[0] = (blockIdx.x == 0) ? f1 : f[j - nx - 1];
    }

    if (threadIdx.x == blockDim.x - 1) {
      fs[threadIdx.x + 2] = (blockIdx.x == gridDim.x - 1) ? f1 : f[j - nx + 1];
    }

    __syncthreads();

    f_tmp[j - nx] = c0 * (fs[threadIdx.x] + fs[threadIdx.x + 2]) +
                     c1 * (f0 + f2) + c2 * f1;

    j += nx;

    f0 = f1;
    f1 = f2;
  }
}

template <typename T>
DiffusionCalculator<T>::DiffusionCalculator(
    const size_t nx, const size_t ny, const T kappa)
  : time_(0), nx_(nx), ny_(ny), n_(nx * ny),
    f_(nx * ny, 0), buf_(nx * ny, 0), initialized_(false) {

  const T dt = 0.20 / (kappa * std::pow(std::max(nx, ny), 2));
  c0_ = kappa * dt * (std::pow(nx, 2));
  c1_ = kappa * dt * (std::pow(ny, 2));
  c2_ = 1.0 - 2.0 * (c0_ + c1_);

  for (auto jy = (decltype(ny))0; jy < ny_; ++jy) {
    for (auto jx = (decltype(nx))0; jx < nx_; ++jx) {
      const auto j = nx_ * jy + jx;
      const auto x = (jx + 0.5) / nx - 0.5;
      const auto y = (jy + 0.5) / ny - 0.5;

      f_[j] = std::exp(-100.0 * (std::pow(x, 2) + std::pow(y, 2)));
    }
  }
}

template <typename T>
bool DiffusionCalculator<T>::Initialize() {
  if (initialized_) return false;

  const auto f_size = n_ * sizeof(T);
  CUDA_SAFE_CALL(hipMalloc((void**)&D_f_, f_size));
  CUDA_SAFE_CALL(hipMalloc((void**)&D_tmp_, f_size));
  CUDA_SAFE_CALL(hipMemcpy((void*)D_f_, (void*)f_.data(), f_size, hipMemcpyHostToDevice));

  initialized_ = true;
  return true;
}

template <typename T>
bool DiffusionCalculator<T>::Compute() {
  if (!initialized_) return false;

  dim3 grid(nx_ / blockDim_x, ny_ / blockDim_y, 1);
  dim3 threads(blockDim_x, 1, 1);
  diffusion2d<<<grid, threads>>>(D_f_, D_tmp_, nx_, ny_, c0_, c1_, c2_);
  std::swap(D_f_, D_tmp_);
  
  time_++;
  return true; 
}

template <typename T>
bool DiffusionCalculator<T>::Dump() {
  if (!initialized_) return false;

  const auto size = nx_ * ny_;
  CUDA_SAFE_CALL(hipMemcpy((void*)f_.data(), (void*)D_f_, size * sizeof(T), hipMemcpyDeviceToHost));

  for (auto i = (decltype(size))0; i < size; ++i) {
    buf_[i] = static_cast<uint8_t>(std::round(255 * f_[i]));
  }

  std::ostringstream sout;
  sout << "results/" << std::setfill('0') << std::setw(6) << time_ << ".bmp";
  return bmp::WriteBmp(sout.str(), nx_, ny_, buf_.data());
}

template <typename T>
bool DiffusionCalculator<T>::Finalize() {
  if (!initialized_) return false;

  CUDA_SAFE_CALL(hipFree((void*)D_f_));
  CUDA_SAFE_CALL(hipFree((void*)D_tmp_));
  initialized_ = false;
  return true;
}

template class DiffusionCalculator<float>;
template class DiffusionCalculator<double>;

} // namespace diffusion 
